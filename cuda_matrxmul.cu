
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <time.h>
using namespace std;

__global__ void GPU_MatMul(float *A, float *B, float *C, int N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0.f;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{	
	cout << "Executing Matrix Multiplcation" << endl;
	for(int BLOCK_SIZE =1; BLOCK_SIZE<=10; BLOCK_SIZE++){
		// Perform matrix multiplication C = A*B
		// where A, B and C are NxN matrices
		// Restricted to matrices where N = K*BLOCK_SIZE;
		int N,K;
		K = 100;		
		N = K*BLOCK_SIZE;
		clock_t t;
		
		t= clock();
		
		cout << "Matrix size: " << N << "x" << N << endl;
	
		// Allocate memory on the host
		float *hA,*hB,*hC;
		hA = new float[N*N];
		hB = new float[N*N];
		hC = new float[N*N];
	
		// Initialize matrices on the host
		for (int j=0; j<N; j++){
			for (int i=0; i<N; i++){
				hA[j*N+i] = 2.f*(j+i);
				hB[j*N+i] = 1.f*(j-i);
			}
		}
	
		// Allocate memory on the device
		int size = N*N*sizeof(float);	// Size of the memory in bytes
		float *dA,*dB,*dC;
		hipMalloc(&dA,size);
		hipMalloc(&dB,size);
		hipMalloc(&dC,size);
	
		dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
		dim3 grid(K,K);
		
		// Copy matrices from the host to device
		hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
		hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
		
		//Execute the matrix multiplication kernel
		
		GPU_MatMul<<<grid,threadBlock>>>(dA,dB,dC,N);
			
		// Now do the matrix multiplication on the CPU
		float sum;
		for (int row=0; row<N; row++){
			for (int col=0; col<N; col++){
				sum = 0.f;
				for (int n=0; n<N; n++){
					sum += hA[row*N+n]*hB[n*N+col];
				}
				hC[row*N+col] = sum;
			}
		}
		
		// Allocate memory to store the GPU answer on the host
		float *C;
		C = new float[N*N];
		
		// Now copy the GPU result back to CPU
		hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);
		
		// Check the result and make sure it is correct
		for (int row=0; row<N; row++){
			for (int col=0; col<N; col++){
				if ( C[row*N+col] != hC[row*N+col] ){
					cout << "Wrong answer!" << endl;
					row = col = N;
				}
			}
		}
		t = clock() - t;
		cout<<"Time taken is: "<<((float)t)/CLOCKS_PER_SEC<<endl;
		cout << "Finished." << endl;
		
	}getchar();
}